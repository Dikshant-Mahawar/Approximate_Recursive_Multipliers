#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__device__ uint16_t exact_4x4_mult(uint8_t a, uint8_t b) {
    a &= 0xF;
    b &= 0xF;
    return static_cast<uint16_t>(a) * static_cast<uint16_t>(b);
}

__device__ unsigned char d_n1_4x4_mult(unsigned char a, unsigned char b) {
    a &= 0xF;
    b &= 0xF;
    unsigned char Y = 0;
    Y |= (a & 1) & (b & 1);
    Y |= ((((a >> 1) & 1) & (b & 1)) | ((a & 1) & ((b >> 1) & 1))) << 1;
    Y |= ((((a >> 2) & 1) & (b & 1)) | (((a >> 1) & 1) & ((b >> 1) & 1)) | ((a & 1) & ((b >> 2) & 1))) << 2;
    Y |= ((((a >> 3) & 1) & (b & 1)) | (((a >> 2) & 1) & ((b >> 1) & 1)) | (((a >> 1) & 1) & ((b >> 2) & 1)) | ((a & 1) & ((b >> 3) & 1))) << 3;
    
    unsigned char a3b1 = ((a >> 3) & 1) & ((b >> 1) & 1);
    unsigned char a2b2 = ((a >> 2) & 1) & ((b >> 2) & 1);
    unsigned char a1b3 = ((a >> 1) & 1) & ((b >> 3) & 1);
    unsigned char a3b2 = ((a >> 3) & 1) & ((b >> 2) & 1);
    unsigned char a2b3 = ((a >> 2) & 1) & ((b >> 3) & 1);
    unsigned char a3b3 = ((a >> 3) & 1) & ((b >> 3) & 1);
    
    unsigned char C_45_1_approx = a2b2 & (a1b3 | a3b1);
    unsigned char C_56_2_approx = a2b2 & (a3b3 | a3b1 | a1b3);
    
    Y |= (a3b1 | a2b2 | a1b3) << 4;
    Y |= (a3b2 ^ a2b3 ^ C_45_1_approx) << 5;
    Y |= ((a3b3 & (!a2b2)) | ((!a3b3) & a2b2 & (a3b1 | a1b3))) << 6;
    Y |= (a2b2 & a3b3) << 7;
    
    return Y;
}

__device__ uint16_t n8_5(uint8_t a, uint8_t b) {
    uint8_t aL = a & 0xF;
    uint8_t aH = (a >> 4) & 0xF;
    uint8_t bL = b & 0xF;
    uint8_t bH = (b >> 4) & 0xF;
    
    uint16_t aL_bL = d_n1_4x4_mult(aL, bL);
    uint16_t aH_bL = exact_4x4_mult(aH, bL);
    uint16_t aL_bH = exact_4x4_mult(aL, bH);
    uint16_t aH_bH = exact_4x4_mult(aH, bH);
    
    return aL_bL + (aH_bL << 4) + (aL_bH << 4) + (aH_bH << 8);
}

__device__ uint32_t n16_5(uint16_t a, uint16_t b) {
    uint8_t aL = a & 0xFF;
    uint8_t aH = (a >> 8) & 0xFF;
    uint8_t bL = b & 0xFF;
    uint8_t bH = (b >> 8) & 0xFF;
    
    uint16_t aL_bL = n8_5(aL, bL);
    uint16_t aH_bL = n8_5(aH, bL);
    uint16_t aL_bH = n8_5(aL, bH);
    uint16_t aH_bH = n8_5(aH, bH);
    
    uint32_t padded_aL_bL = aL_bL;
    uint32_t padded_aH_bL = static_cast<uint32_t>(aH_bL) << 8;
    uint32_t padded_aL_bH = static_cast<uint32_t>(aL_bH) << 8;
    uint32_t padded_aH_bH = static_cast<uint32_t>(aH_bH) << 16;
    
    return padded_aL_bL + padded_aH_bL + padded_aL_bH + padded_aH_bH;
}


/*

CUDA does not natively support atomic operations on double in all architectures

*/


__global__ void validate_multiplier(unsigned long long *correct_results, double *total_error_distance, 
                                    double *total_relative_error, double *total_squared_error,
                                    unsigned long long *total_tests_mred, double max_value) {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned short a = idx >> 16;
    unsigned short b = idx & 0xFFFF;

    unsigned long long expected = (unsigned long long)a * b;
    unsigned long long result = n16_5(a, b);

    double error_distance = fabs(result - expected);
    double relative_error = (expected != 0) ? (error_distance / expected) : 0;
    double squared_error = error_distance * error_distance;

    // Atomic operations to accumulate metrics across threads
    atomicAdd(correct_results, (result == expected) ? 1ULL : 0ULL);
    atomicAdd(total_error_distance, error_distance);
    atomicAdd(total_relative_error, relative_error);
    atomicAdd(total_squared_error, squared_error);
    if (expected != 0) atomicAdd(total_tests_mred, 1ULL);  // Only count non-zero expected values for MRED
}

int main() {
    const unsigned long long total_tests = 65536ULL * 65536ULL;
    const double max_value = 65535 * 65535;

    // Allocate memory for metric accumulators on the device
    unsigned long long *d_correct_results, correct_results = 0;
    double *d_total_error_distance, total_error_distance = 0;
    double *d_total_relative_error, total_relative_error = 0;
    double *d_total_squared_error, total_squared_error = 0;
    unsigned long long *d_total_tests_mred, total_tests_mred = 0;

    hipMalloc((void**)&d_correct_results, sizeof(unsigned long long));
    hipMalloc((void**)&d_total_error_distance, sizeof(double));
    hipMalloc((void**)&d_total_relative_error, sizeof(double));
    hipMalloc((void**)&d_total_squared_error, sizeof(double));
    hipMalloc((void**)&d_total_tests_mred, sizeof(unsigned long long));

    // Initialize device memory to zero
    hipMemcpy(d_correct_results, &correct_results, sizeof(unsigned long long), hipMemcpyHostToDevice);
    hipMemcpy(d_total_error_distance, &total_error_distance, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_total_relative_error, &total_relative_error, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_total_squared_error, &total_squared_error, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_total_tests_mred, &total_tests_mred, sizeof(unsigned long long), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    unsigned long long numBlocks = (total_tests + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    validate_multiplier<<<numBlocks, threadsPerBlock>>>(d_correct_results, d_total_error_distance, 
                                                        d_total_relative_error, d_total_squared_error,
                                                        d_total_tests_mred, max_value);

    // Copy results back to host
    hipMemcpy(&correct_results, d_correct_results, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    hipMemcpy(&total_error_distance, d_total_error_distance, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&total_relative_error, d_total_relative_error, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&total_squared_error, d_total_squared_error, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&total_tests_mred, d_total_tests_mred, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_correct_results);
    hipFree(d_total_error_distance);
    hipFree(d_total_relative_error);
    hipFree(d_total_squared_error);
    hipFree(d_total_tests_mred);

    // Compute final metrics
    double accuracy = (correct_results * 100.0) / total_tests;
    double error_rate = 100.0 - accuracy;
    double nmed = total_error_distance / (total_tests * max_value);
    double mred = total_relative_error / total_tests_mred;
    double noeb = (2 * 8) - log2(1.0 + sqrt(total_squared_error / total_tests));

    // Display results
    std::cout << "=== Performance Metrics ===\n";
    std::cout << "Total tests: " << total_tests << "\n";
    std::cout << "Correct results: " << correct_results << "\n";
    std::cout << "Accuracy: " << accuracy << "%\n";
    std::cout << "Error rate: " << error_rate << "%\n\n";

    std::cout << "=== Error Metrics ===\n";
    std::cout << "Total Error Distance: " << total_error_distance << "\n";
    std::cout << "Total Relative Error: " << total_relative_error << "\n";
    std::cout << "NMED (Normalized Mean Error Distance): " << nmed << "\n";
    std::cout << "MRED (Mean Relative Error Distance): " << mred << "\n";
    std::cout << "NoEB (Number of Effective Bits): " << noeb << "\n";

    return 0;
}
